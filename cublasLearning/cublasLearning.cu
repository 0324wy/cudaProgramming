#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Define matrix dimensions
    int m = 2, n = 2, k = 2;

    // Host matrices
    float h_A[m * k] = {1, 2, 3, 4};
    float h_B[k * n] = {5, 6, 7, 8};
    float h_C[m * n];

    // Device matrices
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, m * k * sizeof(float));
    hipMalloc((void**)&d_B, k * n * sizeof(float));
    hipMalloc((void**)&d_C, m * n * sizeof(float));

    // Transfer data to device
    hipMemcpy(d_A, h_A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, k * n * sizeof(float), hipMemcpyHostToDevice);

    // Perform matrix multiplication: h_C = h_A * h_B
    const float alpha = 1.0f, beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, m, d_B, k, &beta, d_C, m);

    // Copy result back to host
    hipMemcpy(h_C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // Print result
    printf("Result matrix C:\n");
    for (int i = 0; i < m * n; ++i) printf("%f ", h_C[i]);
    printf("\n");

    // Clean up
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    hipblasDestroy(handle);

    return 0;
}