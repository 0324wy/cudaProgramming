#include <hipblaslt.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdint>

int main() {
    hipblasLtHandle_t ltHandle;
    hipblasLtCreate(&ltHandle);

    int M = 128, N = 128, K = 128;

    int32_t h_A[M * K], h_B[K * N], h_C[M * N];

    for (int i = 0; i < M * K; i++) h_A[i] = (i % 100) + 1;
    for (int i = 0; i < K * N; i++) h_B[i] = 1;

    int32_t *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, M * K * sizeof(int32_t));
    hipMalloc((void**)&d_B, K * N * sizeof(int32_t));
    hipMalloc((void**)&d_C, M * N * sizeof(int32_t));

    hipMemcpy(d_A, h_A, M * K * sizeof(int32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(int32_t), hipMemcpyHostToDevice);

    hipblasLtMatmulDesc_t opDesc;
    hipblasLtMatmulDescCreate(&opDesc, HIPBLAS_COMPUTE_32I, HIP_R_32I);

    hipblasOperation_t transA = HIPBLAS_OP_N;
    hipblasOperation_t transB = HIPBLAS_OP_N;
    hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transA, sizeof(transA));
    hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transB, sizeof(transB));

    hipblasLtMatrixLayout_t Adesc, Bdesc, Cdesc;
    hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32I, M, K, K); // Leading dimension K for A
    hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32I, K, N, N); // Leading dimension N for B
    hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32I, M, N, N); // Leading dimension N for C

    int32_t alpha = 1;
    int32_t beta = 0;

    hipblasStatus_t status = hipblasLtMatmul(ltHandle, opDesc, &alpha, d_A, Adesc, d_B, Bdesc, &beta, d_C, Cdesc, d_C, Cdesc, nullptr, nullptr, 0, 0);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLASLt matrix multiplication failed with error code: " << status << std::endl;
        return -1;
    }

    hipDeviceSynchronize(); // Ensure all operations are complete before copying back

    hipMemcpy(h_C, d_C, M * N * sizeof(int32_t), hipMemcpyDeviceToHost);

    for (int i = 0; i < 100; i++) {
        std::cout << h_C[i] << " ";
    }
    std::cout << std::endl;

    hipblasLtMatrixLayoutDestroy(Adesc);
    hipblasLtMatrixLayoutDestroy(Bdesc);
    hipblasLtMatrixLayoutDestroy(Cdesc);
    hipblasLtMatmulDescDestroy(opDesc);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasLtDestroy(ltHandle);

    return 0;
}