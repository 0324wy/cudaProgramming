#include <hipblaslt.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdint>

// Helper function to print __uint128_t as two 64-bit integers
void print_uint128(__uint128_t value) {
    uint64_t high = value >> 64;
    uint64_t low = static_cast<uint64_t>(value);
    std::cout << "High: " << high << ", Low: " << low << std::endl;
}

int main() {
    // Initialize cuBLASLt
    hipblasLtHandle_t ltHandle;
    hipblasLtCreate(&ltHandle);

    // Define matrix dimensions (example: 128x128)
    int M = 128, N = 128, K = 128;

    // Host matrices (int32)
    __uint128_t h_A[M * K], h_B[K * N], h_C[M * N];

    // Initialize matrices with some values (fill h_A and h_B as needed)
    for (int i = 0; i < M * K; i++) h_A[i] = i;    // Populate h_A
    for (int i = 0; i < K * N; i++) h_B[i] = i * 2; // Populate h_B


    // Device matrices
    __uint128_t *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, M * K * sizeof(__uint128_t));
    hipMalloc((void**)&d_B, K * N * sizeof(__uint128_t));
    hipMalloc((void**)&d_C, M * N * sizeof(__uint128_t));

    // Copy data to device
    hipMemcpy(d_A, h_A, M * K * sizeof(__uint128_t), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(__uint128_t), hipMemcpyHostToDevice);

    // Create operation descriptor
    hipblasLtMatmulDesc_t opDesc;
    hipblasLtMatmulDescCreate(&opDesc, HIPBLAS_COMPUTE_32I, HIP_R_32I);  // Corrected: cublasComputeType_t and cudaDataType_t

    // Set matrix operation attributes
    hipblasOperation_t transA = HIPBLAS_OP_N;
    hipblasOperation_t transB = HIPBLAS_OP_N;
    hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transA, sizeof(transA));
    hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transB, sizeof(transB));

    // Create matrix layout descriptors
    hipblasLtMatrixLayout_t Adesc, Bdesc, Cdesc;
    hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32I, M, K, M);  // CUDA_R_32I for int32 data type
    hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32I, K, N, K);
    hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32I, M, N, M);

    // Scaling factors
    __uint128_t alpha = 1;
    __uint128_t beta = 0;

    // Launch integer GEMM
    hipblasLtMatmul(ltHandle, opDesc, &alpha, d_A, Adesc, d_B, Bdesc, &beta, d_C, Cdesc, d_C, Cdesc, nullptr, nullptr, 0, 0);

    // Copy result back to host
    hipMemcpy(h_C, d_C, M * N * sizeof(__uint128_t), hipMemcpyDeviceToHost);

    // Print result (first few elements) using the helper function
    for (int i = 0; i < 10; i++) {
        print_uint128(h_C[i]);
    }

    // Cleanup
    hipblasLtMatrixLayoutDestroy(Adesc);
    hipblasLtMatrixLayoutDestroy(Bdesc);
    hipblasLtMatrixLayoutDestroy(Cdesc);
    hipblasLtMatmulDescDestroy(opDesc);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasLtDestroy(ltHandle);

    return 0;
}