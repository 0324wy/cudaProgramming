#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void matrixVec(float *inVec, float *inMatrix, float *outVec, int width)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < width)
    {
        float outValue = 0;
        for (size_t j = 0; j < width; j++)
        {
            outValue += inVec[j] * inMatrix[i + j * width];
        }
        outVec[i] = outValue;
    }
}

void printArray(float *h_num, int n)
{
    for (size_t i = 0; i < n; i++)
    {
        cout << h_num[i] << endl;
    }
}

int main()
{
    int width = 3;
    float *h_inVec = (float *)malloc(width * sizeof(float));
    float *h_inMatrix = (float *)malloc(width * width * sizeof(float));
    float *h_outVec = (float *)malloc(width * sizeof(float));

    for (size_t i = 0; i < width; i++)
    {
        h_inVec[i] = static_cast<float>(i);
        for (size_t j = 0; j < width; j++)
        {
            h_inMatrix[i * width + j] = static_cast<float>(j);
        }
    }
    
    float *d_inVec;
    float *d_inMatrix;
    float *d_outVec;

    hipMalloc((void **)&d_inVec, width * sizeof(float));
    hipMalloc((void **)&d_inMatrix, width * width * sizeof(float));
    hipMalloc((void **)&d_outVec, width * sizeof(float));

    hipMemcpy(d_inVec, h_inVec, width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_inMatrix, h_inMatrix, width * width * sizeof(float), hipMemcpyHostToDevice);

    matrixVec<<<ceil(width / 64.0), 64>>>(d_inVec, d_inMatrix, d_outVec, width);


    hipMemcpy(h_outVec, d_outVec, width * sizeof(float), hipMemcpyDeviceToHost);

    printArray(h_outVec, width);

    hipFree(d_inVec);
    hipFree(d_inMatrix);
    hipFree(d_outVec);

    free(h_inVec);
    free(h_inMatrix);
    free(h_outVec);

    return 0;
}