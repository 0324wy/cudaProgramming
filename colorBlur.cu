#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>

using namespace std;
using namespace cv;

#define BlurSize 4

__global__ void colorBlur(unsigned char *pin, unsigned char *pout, int width, int height)
{
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int row = threadIdx.y + blockDim.y * blockIdx.y;

    if (row < height && col < width)
    {
        for (int channel = 0; channel < 3; ++channel) {  // Loop over RGB channels
            int pixVal = 0;
            int pixels = 0;

            for (int blurRow = -BlurSize; blurRow <= BlurSize; ++blurRow)
            {
                for (int blurCol = -BlurSize; blurCol <= BlurSize; ++blurCol)
                {
                    int curRow = row + blurRow;
                    int curCol = col + blurCol;

                    // Ensure the neighboring pixel is within the image boundaries
                    if (curRow >= 0 && curRow < height && curCol >= 0 && curCol < width)
                    {
                        // Access the correct channel (R, G, or B)
                        pixVal += pin[(curRow * width + curCol) * 3 + channel];  
                        pixels++;
                    }
                }
            }

            // Write the averaged pixel value to the output for the current channel
            pout[(row * width + col) * 3 + channel] = (unsigned char)(pixVal / pixels);
        }
    }
}

int main()
{
    // Load the image using OpenCV
    Mat image = imread("./image.jpeg");
    if (image.empty()) {
        cout << "Error: Could not load image!" << endl;
        return -1;
    }

    int width = image.cols;
    int height = image.rows;

    // Calculate the size of the image in bytes (3 channels)
    int img_size = width * height * 3 * sizeof(unsigned char);

    // Host input and output pointers
    unsigned char *h_pin = image.data;
    unsigned char *h_pout = (unsigned char *)malloc(img_size);

    // Device input and output pointers
    unsigned char *d_pin, *d_pout;
    hipMalloc((void **)&d_pin, img_size);
    hipMalloc((void **)&d_pout, img_size);

    // Copy the input image to the GPU
    hipMemcpy(d_pin, h_pin, img_size, hipMemcpyHostToDevice);

    // Configure the kernel launch with 16x16 threads per block
    dim3 threadPerBlock(16, 16);
    dim3 blockPerGrid((width + threadPerBlock.x - 1) / threadPerBlock.x,
                      (height + threadPerBlock.y - 1) / threadPerBlock.y);

    // Launch the kernel
    colorBlur<<<blockPerGrid, threadPerBlock>>>(d_pin, d_pout, width, height);

    // Copy the result back to the CPU
    hipMemcpy(h_pout, d_pout, img_size, hipMemcpyDeviceToHost);

    // Create the output image
    Mat blurImg(height, width, CV_8UC3, h_pout);

    // Save the result
    imwrite("blurred_image.png", blurImg);

    // Free device and host memory
    hipFree(d_pin);
    hipFree(d_pout);
    free(h_pout);

    return 0;
}